#include "hip/hip_runtime.h"
#include "cufunc.h"
#include <string.h>
#include "mrc.h"
#include "func.h"
#include <signal.h> 
#define PI 3.141592653589793
#define BLOCKSIZE 1024


static __device__ hipComplex conj(hipComplex f)
{
	f.y*=-1.0;
	return f;
}


static __device__ hipComplex cXc(hipComplex a, hipComplex b) // a*b
{
	hipComplex c;
	c.x=a.x*b.x-a.y*b.y;
	c.y=a.x*b.y+a.y*b.x;
	return c;
}

static __device__ float cabs(hipComplex a) // a*b
{
	return sqrt(a.x*a.x+a.y*a.y);
}

bool initGPU(int GPUNum)
{
	//initional CUDA device
	int ngpu;
	hipGetDeviceCount(&ngpu);
	if(ngpu <= 0)
	{
		return false;
	}
	if(GPUNum>=ngpu)
	{
		printf("GPU ID %d is out of range(%d). Abort.\n",GPUNum,ngpu);
		return false;
	}

	hipDeviceProp_t prop;
	if(hipGetDeviceProperties(&prop, GPUNum) == hipSuccess) 
	{
		printf("Use GPU: #%d %s\n",GPUNum,prop.name);
		if(prop.kernelExecTimeoutEnabled)
		{
			printf("Warnning: This GPU is also used for display, may not stable.\n");
		}
	}


	if(hipSetDevice(GPUNum)!=hipSuccess)
	{
		printf("Error: Failed to set CUDA Device #%d. Abort.\n",GPUNum);
		return false;
	}
	
	signal(SIGINT, siginthandler);

	return true;
}

bool ResetGPU()
{
	if(hipDeviceReset()!=hipSuccess) return false;
	return true;
}

void siginthandler(int param) 
{   
	if(ResetGPU()) printf(" GPU was reset successfully after process was killed.\n"); 
	else printf(" Error: Failed to reset GPU.\n"); 
	exit(1); 
}

int getGPUList(vector<string> &namelist)
{
	int ngpu;
	hipGetDeviceCount(&ngpu);
	if(ngpu <= 0)
	{
		return 0;
	}

	namelist.clear();
	int i;
	string str;
	for(i = 0; i < ngpu; i++) 
	{
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) 
		{
			str=prop.name;
			namelist.push_back(str);
		}
	}
	
	return ngpu;
}

void GPUMemCheck(size_t &theFree, size_t &theTotal)
{
	hipMemGetInfo( &theFree, &theTotal );  
	//printf( "CARD returns:  free:%d  total:%d\n", theFree, theTotal);
}

bool GPUMemAlloc(void **buf, int size)
{
	if(hipMalloc((void **)buf,size)!=hipSuccess) return false;
	return true;
}

bool GPUMemZero(void **buf, int size)
{
	if(hipMemset(*buf,0,size)!=hipSuccess) return false;
	return true;
}

bool GPUMemFree(void **buf)
{
	if(hipFree(*buf)!=hipSuccess) return false;
	*buf=0;
	return true;
}

bool GPUMemH2D(void *dst, void *src, int size)
{
	if(hipMemcpy(dst,src,size,hipMemcpyHostToDevice)!=hipSuccess) return false;
	return true;
}

bool GPUMemD2H(void *dst, void *src, int size)
{
	if(hipMemcpy(dst,src,size,hipMemcpyDeviceToHost)!=hipSuccess) return false;
	return true;
}
bool GPUMemD2D(void *dst, void *src, int size)
{
	if(hipMemcpy(dst,src,size,hipMemcpyDeviceToDevice)!=hipSuccess) return false;
	return true;
}

bool GPUCrop2d(float *src, DIM nsamin, float *dst, DIM offset, DIM nsamout)
{
	if(src==0 || dst==0) return false;
	if(nsamin==0 || nsamout==0) return false;
   if((offset+nsamout) > nsamin) return false;

   int nsaminb=nsamin.width()+2;
   int nsamoutb=nsamout.width()+2;

	int i;
	for(i=0;i<nsamout.height();i++)
	{
		GPUMemD2D(dst+i*nsamoutb, src+(i+offset.y)*nsaminb+offset.x, 
				sizeof(float)*nsamout.width());
	}

	return true;

}

bool GPUMemBinD2H(float *dst, float *src, DIM dst_nsam, DIM src_nsam)
{
	int i;
	int size=sizeof(float)*(dst_nsam.x+2);
	for(i=0;i<dst_nsam.y/2;i++)
	{
		//up half
		if(hipMemcpy(dst+i*(dst_nsam.x+2),src+i*(src_nsam.x+2),size,hipMemcpyDeviceToHost)!=hipSuccess) return false;

		//down half
		if(hipMemcpy(dst+(dst_nsam.y-1-i)*(dst_nsam.x+2),src+(src_nsam.y-1-i)*(src_nsam.x+2),size,hipMemcpyDeviceToHost)!=hipSuccess) return false;
	}

	return true;
}
bool GPUMemBinD2D(float *dst, float *src, DIM dst_nsam, DIM src_nsam)
{
	int i;
	int size=sizeof(float)*(dst_nsam.x+2);
	for(i=0;i<dst_nsam.y/2;i++)
	{
		//up half
		if(hipMemcpy(dst+i*(dst_nsam.x+2),src+i*(src_nsam.x+2),size,hipMemcpyDeviceToDevice)!=hipSuccess) return false;

		//down half
		if(hipMemcpy(dst+(dst_nsam.y-1-i)*(dst_nsam.x+2),src+(src_nsam.y-1-i)*(src_nsam.x+2),size,hipMemcpyDeviceToDevice)!=hipSuccess) return false;
	}

	return true;
}

__global__ void cuFFTLogModulus(float *dMod, hipComplex *dfft, int size, float scale)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dMod[id]=log(1+cabs(dfft[id])*scale);
}
void GPUFFTLogModulus(float *dMod, float *dfft, DIM nsam, float scale)
{
	int size=(nsam.width()/2+1)*nsam.height();
	cuFFTLogModulus<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dMod,(hipComplex *)dfft,size,scale);
}

__global__ void cuFFTModulus(float *dMod, hipComplex *dfft, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dMod[id]=cabs(dfft[id]);
}
void GPUFFTModulus(float *dMod, float *dfft, DIM nsam)
{
	int size=(nsam.width()/2+1)*nsam.height();
	cuFFTModulus<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dMod,(hipComplex *)dfft,size);
}

DIM GPURectFFTLogModulus(float *dfft, float *dsum, float *dtmp1, float *dtmp2, DIM nsam, float scale, hipfftHandle hfft)
{
	DIM nsamsub=nsam.MinSquare();
	DIM offset=nsam.MinSquareOffset();

	//crop (0,0) square
	GPUCrop2d(dsum, nsam, dtmp1, DIM(0,0), nsamsub);
	//GPUbuf2mrc("output.mrc",dtmp1,nsamsub.x+2,nsamsub.y,1);
	GPUFFT2d(dtmp1, hfft);
	GPUFFTLogModulus(dfft, dtmp1, nsamsub, scale);

	//crop (offset) square
	//if(offset!=0)
	while(offset!=0)
	{
		GPUCrop2d(dsum, nsam, dtmp1, offset, nsamsub);
		GPUFFT2d(dtmp1, hfft);
		GPUFFTLogModulus(dtmp2, dtmp1, nsamsub, scale);
	
		//add
		GPUAdd(dfft, dtmp2, sizeof(float)*(nsamsub.width()/2+1)*nsamsub.height());
		GPUSync();
		
		offset.x-=nsamsub.x;
		offset.y-=nsamsub.y;
		if(offset.x<0) offset.x=0;
		if(offset.y<0) offset.y=0;
	}
	
	//make pwr
	/*float *buf1=new float[(nsamsub.width()/2+1)*nsamsub.height()];
	float *buf2=new float[(nsamsub.width()+2)*nsamsub.height()];
	GPUMemD2H(buf1,dfft,sizeof(float)*(nsamsub.width()/2+1)*nsamsub.height());
	FFTModulusToDispBuf(buf2, buf1, nsamsub);
	GPUMemH2D(dtmp1,buf2,sizeof(float)*(nsamsub.width()+2)*nsamsub.height());
	printf("here 1");
	//bin to dispdim
	GPUFFT2d(dtmp1, hfft);
	GPUSync();
	GPUMemBinD2D(dfft, dtmp1, DIM(dispdim,dispdim), nsamsub);
	GPUSync();
	//need ifft to final, was done in output function
	printf("here 2");
	delete [] buf1;
	delete [] buf2;*/

	return nsamsub;
}


void GPUBinFFT(float *dst, int dispdim, float *src, DIM nsamsub, hipfftHandle hfft)
{
	//bin to dispdim
	GPUFFT2d(src, hfft);
	GPUMemBinD2D(dst, src, DIM(dispdim,dispdim), nsamsub);
	//need ifft to final, was done in output function
}

void GPUFFTErrorMessage(hipfftResult r, char *name)
{
	char err[10][32]=
	  { "Success 0",  
 		 "InvalidPlan 1",  
 		 "AllocFailed 2",  
 		 "InvalidType 3",  
 		 "InvalidValue 4",  
 		 "InternalError 5",  
 		 "ExecFailed 6",  
 		 "SetupFailed 7",  
  		 "InvalidSize 8",  
 		 "UnalignedData 9"};
 	if(r!=0)	printf("\n\nError: cuFFT error: %s  in function: %s\n\n",err[r],name);
}

hipfftHandle GPUFFTPlan(DIM nsam)
{
	hipfftHandle plan;
	hipfftResult r=hipfftPlan2d(&plan,nsam.y,nsam.x,HIPFFT_R2C);
	//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_PADDING);
	GPUFFTErrorMessage(r,"GPUFFTPlan");
	return plan;
}
hipfftHandle GPUIFFTPlan(DIM nsam)
{
	hipfftHandle plan;
	hipfftResult r=hipfftPlan2d(&plan,nsam.y,nsam.x,HIPFFT_C2R);
	//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_PADDING);
	GPUFFTErrorMessage(r,"GPUIFFTPlan");
	return plan;
}

void GPUFFTDestroy(hipfftHandle &plan)
{
	hipfftDestroy(plan);
	plan=0;
}

bool GPUFFT2d(float* dfft, hipfftHandle plan)
{
	//if(hipfftExecR2C(plan,(hipfftReal*)dfft,(hipfftComplex *)dfft)!=HIPFFT_SUCCESS) return false;
	hipfftResult r=hipfftExecR2C(plan,(hipfftReal*)dfft,(hipfftComplex *)dfft);
	GPUFFTErrorMessage(r,"GPUFFT2d");
	if(r!=HIPFFT_SUCCESS) return false;
	return true;
}

bool GPUIFFT2d(float* dfft, hipfftHandle plan)
{
	//if(hipfftExecC2R(plan,(hipfftComplex *)dfft,(hipfftReal*)dfft)!=HIPFFT_SUCCESS) return false;
	hipfftResult r=hipfftExecC2R(plan,(hipfftComplex *)dfft,(hipfftReal*)dfft);
	GPUFFTErrorMessage(r,"GPUIFFT2d");
	if(r!=HIPFFT_SUCCESS) return false;
	return true;
}

bool GPUSync()
{
	if(hipDeviceSynchronize()!=hipSuccess) return false;
	if(hipGetLastError()!=hipSuccess) return false;
	return true;
}


__global__ void cuAdd(float *dst, float *src, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dst[id]+=src[id];
}
void GPUAdd(float *dst, float *src, int size)
{
	cuAdd<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dst,src,size);
}

__global__ void cuMultiplyNum(float *dst, float num, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dst[id]*=num;
}
void GPUMultiplyNum(float *dst, float num, int size)
{
	cuMultiplyNum<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dst,num,size);
}


//int3[x,y,sign]
/*void MkPosList(int3 *list, int nsam, float inner_r, float outer_r)
{
	int hnsamb=nsam/2+1;
	int hnsam=nsam/2;
	int i,j;
	int count=0;
	int r2;
	int ri2=int(inner_r*inner_r);
	int ro2=int(outer_r*outer_r);
	for(j=0;j<hnsam;j++)
		for(i=0;i<hnsamb;i++)
		{
			list[count].x=i;
			list[count].y=j;

			r2=list[count].x*list[count].x+list[count].y*list[count].y;
			if(r2<ri2 || r2>ro2) list[count].z=0;
			else 
			{
				if((list[count].x+list[count].y)%2==0) list[count].z=1;
				else list[count].z=-1;
			}

			count++;
		}
	for(j=hnsam;j<nsam;j++)
		for(i=0;i<hnsamb;i++)
		{
			list[count].x=i;
			list[count].y=j-nsam;
			r2=list[count].x*list[count].x+list[count].y*list[count].y;
			if(r2<ri2 || r2>ro2) list[count].z=0;
			else 
			{
				if((list[count].x+list[count].y)%2==0) list[count].z=1;
				else list[count].z=-1;
			}
			count++;
		}

}*/

//MASK[x,y,sign*bfactor]
void MkPosList(MASK *list, DIM nsam, float bfactor)
{
	int hnsamxb=nsam.x/2+1;
	int hnsamy=nsam.y/2;
	int i,j;
	int count=0;
	float r2;
	float m=-0.5*bfactor;
	DIM nsam2=nsam*nsam;
	
	for(j=0;j<hnsamy;j++)
		for(i=0;i<hnsamxb;i++)
		{
			list[count].x=i;
			list[count].y=j;

			r2=list[count].x*list[count].x/float(nsam2.x)+list[count].y*list[count].y/float(nsam2.y);
			if((list[count].x+list[count].y)%2==0) list[count].z=exp(m*r2);
			else list[count].z=-exp(m*r2);

			count++;
		}
	for(j=hnsamy;j<nsam.y;j++)
		for(i=0;i<hnsamxb;i++)
		{
			list[count].x=i;
			list[count].y=j-nsam.height();
			r2=list[count].x*list[count].x/float(nsam2.x)+list[count].y*list[count].y/float(nsam2.y);
			if((list[count].x+list[count].y)%2==0) list[count].z=exp(m*r2);
			else list[count].z=-exp(m*r2);

			count++;
		}

}

__global__ void cuShiftCC(hipComplex *dfft, hipComplex *dsum, MASK *dposlist,float shx, float shy, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;

	MASK pos=dposlist[id];
	float shift=shx*pos.x+shy*pos.y;
	hipComplex phase;
	phase.x=cos(shift);
	phase.y=sin(shift);

	hipComplex val=cXc(dsum[id],conj(cXc(dfft[id],phase)));
	val.x/=size;
	val.x*=pos.z;
	val.y/=size;
	val.y*=pos.z;
	dfft[id]=val;

}
void GPUShiftCC(float *dfft, float *dsum, MASK *dposlist, float sx, float sy, DIM nsam)
{
	float shx=sx*2.0*PI/nsam.x;
	float shy=sy*2.0*PI/nsam.y;

	int size=(nsam.width()/2+1)*nsam.height();
	cuShiftCC<<<size/BLOCKSIZE+1,BLOCKSIZE>>>((hipComplex *)dfft, (hipComplex *)dsum, dposlist,shx, shy, size);

}

__global__ void cuShift(hipComplex *dfft,MASK *dposlist,float shx, float shy, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;

	MASK pos=dposlist[id];
	float shift=shx*pos.x+shy*pos.y;
	hipComplex phase;
	phase.x=cos(shift);
	phase.y=sin(shift);

	dfft[id]=cXc(dfft[id],phase);

}
void GPUShift(float *dfft, MASK *dposlist, float sx, float sy, DIM nsam)
{
	float shx=sx*2.0*PI/nsam.x;
	float shy=sy*2.0*PI/nsam.y;

	int size=(nsam.width()/2+1)*nsam.height();
	cuShift<<<size/BLOCKSIZE+1,BLOCKSIZE>>>((hipComplex *)dfft,dposlist,shx, shy, size);

}

float FindShift(float *dsrc,DIM nsam, float* hboxmap, int box, float &sx, float &sy, int wNoise)
{
	int orix=(nsam.x-box)/2;
	int oriy=(nsam.y-box)/2;
	int nsamxb=nsam.x+2;
	int i,j,id,is;
	float bestcc=-1e9;

	//float *dst=new float[box*box];
	for(i=0;i<box;i++)
	{
		hipMemcpy(hboxmap+i*box,dsrc+(i+oriy)*nsamxb+orix,sizeof(float)*box,hipMemcpyDeviceToHost);
	}

	sx=0;
	sy=0;
	for(j=0;j<box;j++)
		for(i=0;i<box;i++)
		{
			id=j*box+i;

			if(abs(i-box/2)<=wNoise && abs(j-box/2)<=wNoise) continue;

			if(hboxmap[id]>bestcc)
			{
				bestcc=hboxmap[id];
				sx=i;
				sy=j;
			}
		}


	//Fourier interpolation
	int subbox=16; //box/4;
	float *hsubboxmap=new float[(subbox+2)*subbox];
	int offsetx=int(sx-subbox/2);
	int offsety=int(sy-subbox/2);
	if((offsetx+subbox)>box) offsetx=box-subbox;
	else if(offsetx<0) offsetx=0;
	if((offsety+subbox)>box) offsety=box-subbox;
	else if(offsety<0) offsety=0;
	//crop and fft
	crop2fft(hboxmap,DIM(box,box),hsubboxmap,offsetx,offsety,DIM(subbox,subbox));
	fft2d(hsubboxmap,DIM(subbox,subbox));
	cosmask2d((complex<float> *)hsubboxmap,subbox);
	//pad
	int scale=32;
	int wNoiseScaled=scale*wNoise;
	int pad=subbox*scale;
	float *hpadmap=new float[(pad+2)*pad];
	memset(hpadmap,0,sizeof(float)*(pad+2)*pad);
	for(i=0;i<subbox/2;i++)
	{
		memcpy(hpadmap+i*(pad+2),hsubboxmap+i*(subbox+2),sizeof(float)*(subbox+2));
	}
	for(i=0;i<subbox/2;i++)
	{
		memcpy(hpadmap+(pad-1-i)*(pad+2),hsubboxmap+(subbox-1-i)*(subbox+2),sizeof(float)*(subbox+2));
	}
	//ifft
	ifft2d(hpadmap,DIM(pad,pad));
	//find shift
	int ox=(box/2-offsetx)*scale; //in order to avoid noise peak at box/2
	int oy=(box/2-offsety)*scale; //in order to avoid noise peak at box/2
	int sxp=0,syp=0;
	bestcc=hpadmap[0];
	for(j=0;j<pad;j++)
	{
		is=j*(pad+2);
		for(i=0;i<pad;i++)
		{
			if(abs(i-ox)<=wNoiseScaled && abs(j-oy)<=wNoiseScaled) continue;

			id=is+i;
			if(hpadmap[id]>bestcc)
			{
				bestcc=hpadmap[id];
				sxp=i;
				syp=j;
			}
		}
	}

	sx=offsetx+sxp/double(scale);
	sy=offsety+syp/double(scale);
	sx-=box/2;
	sy-=box/2;

	/*char filename[256];
	sprintf(filename,"D:\\UCSFImage\\DoseFragProcess\\data\\temp.mrc");
	MRC mrc;
	mrc.open(filename,"wb");
	mrc.createMRC(hpadmap,pad+2,pad,1);
	mrc.close();*/



	delete [] hsubboxmap;
	delete [] hpadmap;
	

	return bestcc/nsam.x/nsam.y/subbox/subbox;
}


void GPUbuf2mrc(const char *filename, float* dbuf, int nx, int ny, int nz)
{
	float *buf=new float[nx*ny*nz];
	GPUMemD2H(buf,dbuf,sizeof(float)*nx*ny*nz);
	MRC mrc;
	mrc.open(filename,"wb");
	mrc.createMRC(buf,nx,ny,nz);
	mrc.close();
	
	delete [] buf;
	
	printf("**********buf was written to %s\n", filename);
}

void testCUFFT()
{
	/*int i,j;
	int nsam=26;
	int size=(nsam+2)*nsam;
	float *h=new float[size];
	float *r=new float[size];
	memset(h,0,size*sizeof(float));
	memset(r,0,size*sizeof(float));
	for(j=0;j<nsam;j++)
		for(i=0;i<nsam;i++)
		{
			h[j*(nsam+2)+i]=i+1;
		}

	float *d=0;
	
	GPUMemAlloc((void **)&d,size*sizeof(float));
	GPUMemH2D(d,h,size*sizeof(float));

	hipfftHandle fft_plan=GPUFFTPlan(nsam);
	hipfftHandle ifft_plan=GPUIFFTPlan(nsam);

	GPUFFT2d(d,fft_plan);
	GPUSync();
	GPUIFFT2d(d,ifft_plan);
	GPUSync();

	GPUMultiplyNum(d,1.0/nsam/nsam,size);
	GPUMemD2H(r,d,sizeof(float)*size);

	char hstr[65536]="";
	char rstr[65536]="";
	char str[16]="";
	for(j=0;j<nsam;j++)
	{
		strcat(hstr,"\n");
		strcat(rstr,"\n");
		for(i=0;i<(nsam+2);i++)
		{
			sprintf(str,"%6.3f ",h[j*(nsam+2)+i]);
			strcat(hstr,str);
			sprintf(str,"%6.3f ",r[j*(nsam+2)+i]);
			strcat(rstr,str);
		}
		
	}
	
	GPUMemFree((void **)&d);
	GPUFFTDestroy(fft_plan);
	GPUFFTDestroy(ifft_plan);
	delete [] h;
	delete [] r;*/
	
/*GPUFFT2d(d,fft_plan);
	GPUSync();
	GPUIFFT2d(d,ifft_plan);
	GPUSync();*/
	return;
}
